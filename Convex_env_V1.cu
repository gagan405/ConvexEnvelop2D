#include "hip/hip_runtime.h"
/*********************************************************************************************
		Convex Envelope computation in a 2 dimensional plane
		Version 1.0
Gagan Bihari MIshra					Chiranjeeb Basak

Advisor : Dr. Mireille Gregoire
Institute of Parallel and Distributed Systems
University of Stuttgart, Germany	

**********************************************************************************************/

//NPOINTS points in 2D with known coordinates x,y
//hypothesis: no 3 points are aligned
// we want to calculate the convex envelope of those points
//N.B.: there are are algorithms that are much more efficient than this one,
//but that are also more complicated to parallelize

#include "cutil.h"
#include <stdio.h>

#define NPOINTS 2048 		//number of points
#define TYPE int

#define blockSizeX 256
#define blockSizeY 2


TYPE h_x[NPOINTS];		//x coordinate of points
TYPE h_y[NPOINTS];		//y coordinate of points
int h_edges[NPOINTS*2];		//valid edges
int h_edges_tmp[NPOINTS*2];	//valid edges
int h_res_device[NPOINTS+1];
int h_res[NPOINTS+1];		//result: indexes in order of the different points of the convex envelope
				//can be as long as NPOINTS (the points form a convex polygon)

//there are either 0 or 2 valid edges starting from a point i
//if 0 -> both values at position 2*i and 2*i+1 are -1
//if 2-> the indexes of the other ends from the 2 edges

/****************************


GPU code begins here --


*****************************/

__global__ void find_edges_on_device(TYPE * h_x, TYPE * h_y, int *h_edges){
	
	for (int i=threadIdx.x+blockIdx.x*blockDim.x; i<NPOINTS; i+=blockDim.x*gridDim.x){
		for (int j=threadIdx.y+blockIdx.y*blockDim.y; j<NPOINTS; j+=blockDim.y*gridDim.y)
		{
			if (i>=j)
			{
			continue;  //edge i,j == edge j,i
			}
		
			//all the others points should be on the same side of the edge i,j
			
			//normal to the edge (unnormalized)

			TYPE nx= - ( h_y[j]- h_y[i]);
			TYPE ny= h_x[j]- h_x[i];
		
			int k=0;
			while ((k==i)||(k==j))
			{
				k++;
			} //k will be 0,1,or 2, but different from i and j to avoid 
			
			long scalarProduct=nx* (h_x[k]-h_x[i])+ny* (h_y[k]-h_y[i]);
		
			if (scalarProduct<0)
			{
				nx*=-1;
				ny*=-1;
			}

			//we have now at least one point with scalarProduct>0
			//all the other points should comply with the same condition for
			//the edge to be valid
		
			bool isValid=true;

			//loop on all the points 

			for (int k=0; k<NPOINTS; k++)
			{
				scalarProduct=nx* (h_x[k]-h_x[i])+ny* (h_y[k]-h_y[i]);
				if (scalarProduct <0)
				{	//invalid edge
					isValid = false;
					break;
				}
			}
			
			if (isValid)
			{
				int tmp_i = i;
				int tmp_j = j;

			//we use atomic functions to write to the global memory
			//as two threads might be writing to the same location

				if( -1 != atomicCAS(&h_edges[2*i], -1, tmp_j) )
					h_edges[2*i+1]=j;

				if( -1 != atomicCAS(&h_edges[2*j], -1, tmp_i) )
					h_edges[2*j+1]=i;
			}
		}
	}
}


/****************************


Host code begins here --


*****************************/


void find_edges_on_host(TYPE * h_x, TYPE * h_y, int *h_edges)
{
	//loop on all possible edges == pairs of points
	for (int i=0; i<NPOINTS; i++)
	{
		for (int j=0; j<NPOINTS; j++)
		{
			if (i>=j)
			{
			continue;  //edge i,j == edge j,i
			}
		
			//all the others points should be on the same side of the edge i,j
			
			//normal to the edge (unnormalized)
			TYPE nx= - ( h_y[j]- h_y[i]);
			TYPE ny= h_x[j]- h_x[i];
		
			int k=0;
			while ((k==i)||(k==j))
			{
				k++;
			} //k will be 0,1,or 2, but different from i and j to avoid 
			//scalarProduct=0
		
			TYPE scalarProduct=nx* (h_x[k]-h_x[i])+ny* (h_y[k]-h_y[i]);
			if (scalarProduct<0)
			{
				nx*=-1;
				ny*=-1;
			}
			//we have now at least one point with scalarProduct>0
			//all the other points should comply with the same condition for
			//the edge to be valid
		
			bool isValid=true;
			//loop on all the points 
			for (int k=0; k<NPOINTS; k++)
			{
				scalarProduct=nx* (h_x[k]-h_x[i])+ny* (h_y[k]-h_y[i]);
				if (scalarProduct <0)
				{	//invalid edge
					isValid = false;
					break;
				}
			}
			
			if (isValid)
			{
				//write the edge to h_edges in the first available position
				if (h_edges[2*i]==-1)
				{
					h_edges[2*i]=j;// atomic
					//do a check if the value is updated

				}
				else
				{
					h_edges[2*i+1]=j;
				}
				
				//we write the edge two times for a direct access in the next stage
				if (h_edges[2*j]==-1)
				{
					h_edges[2*j]=i;
				}
				else
				{
					h_edges[2*j+1]=i;
				}
				
			}
		}
	}
	return;
}

/****************************

GPU calculation can result in an array which is inverted version of the host result.
In such a case, we have to revert the array before comparing with the CPU result.

*****************************/

void reverse_array(int *h_res_device, int *reversed_dev_result)
{	
	int i = NPOINTS;
	int j = 0;
	
	while(h_res_device[i] == -1){
		reversed_dev_result[i] = -1;
		i--;
	}
	while(i >= 0){
		reversed_dev_result[j] = h_res_device[i];
		j++;
		i--;
	}

}
/****************************

This function sorts the array returned by the host/GPU function and finds out the 
points which form the convex polygon

*****************************/
int sort_edges(int * h_edges, int * h_res)
{
	// find the first point that belongs to the convex envelope
	int i0=0;
	int lastValue = 0;
	while ((i0<NPOINTS)&&(h_edges[2*i0]==-1))
	{
		i0++;
	}
	
	//i0 now belongs to the envelope
	
	h_res[0]=i0;
	h_res[1]=h_edges[2*i0];
	int k=2; //index in the envelope
	lastValue=h_res[1];
	while (k<=NPOINTS)
	{
		//follow the edges, take the points that are not already in h_res
		if (h_edges[2*lastValue]==h_res[k-2])
		{
			h_res[k]=h_edges[2*lastValue+1];
		}
		else
		{
			h_res[k]=h_edges[2*lastValue];
		}
		lastValue=h_res[k];
		if (h_res[k]==i0)
		{
			break;			
		}
		k++;
	}
	
	return k;

}

/****************************

This function compares two arrays passed as arguements

*****************************/
bool check(int *a,int* b){
  for(int i=0;i<NPOINTS+1;i++){
        if(a[i]!=b[i]){
    		return true;
   	}
  }
  return false;
}

/****************************


The main function


*****************************/


int main(int argc, char** args  )
{

  TYPE *dX;
  TYPE *dY;
  int *d_edges;
  int *reversed_dev_result;

//we need timers to measure the execution times

  unsigned int timer1=0;
  unsigned int timer2=0;

//allocate memory in the device

  hipMalloc((void **)&dX,NPOINTS*sizeof(TYPE));
  hipMalloc((void **)&dY,NPOINTS*sizeof(TYPE));
  hipMalloc((void **)&d_edges,NPOINTS*2*sizeof(int));

  CUT_SAFE_CALL(cutCreateTimer(&timer1));
  CUT_SAFE_CALL(cutCreateTimer(&timer2));

//initialisation of the coordinates

for (int i=0; i<NPOINTS; i++)//loop on points
{
	h_x[i]=(rand()%100000);//10000.0f-0.5f;
	h_y[i]=(rand()%100000);//10000.0f-0.5f;
	h_edges[i*2]=-1; 		//no valid edge by default
	h_edges[i*2+1]=-1;
	h_edges_tmp[i*2]=-1; 		//no valid edge by default
	h_edges_tmp[i*2+1]=-1; 
    	h_res[i+1]=-1;
	h_res_device[i+1]=-1;
	
}


//Copy the arrays from the host to the device

  hipMemcpy(dX,h_x,NPOINTS*sizeof(TYPE),hipMemcpyHostToDevice);  
  hipMemcpy(dY,h_y,NPOINTS*sizeof(TYPE),hipMemcpyHostToDevice);  
  hipMemcpy(d_edges,h_edges_tmp,NPOINTS*2*sizeof(int),hipMemcpyHostToDevice);  

//Set the grid and block dimensions before calling the kernel

  dim3 bS(blockSizeX,blockSizeY,1);		
  dim3 gS((NPOINTS+blockSizeX-1)/blockSizeX,(NPOINTS+blockSizeY-1)/blockSizeY,1);

//Start the timer
  CUT_SAFE_CALL(cutStartTimer(timer1));

//Call the CUDA Kernel
  find_edges_on_device<<<gS,bS>>>(dX,dY,d_edges);
  hipDeviceSynchronize();

//Stop the timer
  CUT_SAFE_CALL(cutStopTimer(timer1));

//Copy the result back to the Host
  hipMemcpy(h_edges_tmp,d_edges,NPOINTS*2*sizeof(int),hipMemcpyDeviceToHost);  

//find out which edges belong to the convex envelope in the host function
CUT_SAFE_CALL(cutStartTimer(timer2));

find_edges_on_host(h_x, h_y, h_edges);

CUT_SAFE_CALL(cutStopTimer(timer2));

float time1=cutGetAverageTimerValue(timer1);
float time2=cutGetAverageTimerValue(timer2);
printf(" Time on Device %f ms \t  Time on Host %f ms\n",time1, time2);

//reset timer1 and timer2
CUT_SAFE_CALL(cutResetTimer(timer1));
CUT_SAFE_CALL(cutResetTimer(timer2));

//sort edges found in host
int nedges=sort_edges(h_edges, h_res);

//sort edges found in device

nedges=sort_edges(h_edges_tmp, h_res_device);

//compare the host and device results

if(check(h_res,h_res_device)){
	
	//if they mismatch, possibly the device result is the reverse of host result
	//in such a case, reverse one of the array and compare again

	reversed_dev_result = (int *)malloc((NPOINTS+1) * sizeof(int));
	reverse_array(h_res_device, reversed_dev_result);
	
	//after reversing, compare again

	if(check(h_res,reversed_dev_result))
		printf("\nError!\n");
	else
		printf("\nPassed!\n");
	
	//Don't forget to free the allocated memory
	free(reversed_dev_result);
  }
  else
    printf("\nPassed!\n");

//free the allocated memory in device

hipFree(dX);  
hipFree(dY);  
hipFree(d_edges);  

return 0;
}
